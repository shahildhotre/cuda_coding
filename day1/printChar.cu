#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

__constant__ char d_message[40];

__global__ void msgRead(char* d_msg)
{
    int id = blockIdx.x*blockDim.x + threadIdx.x;

    d_msg[id] = d_message[id];
}



int main(){

    char* h_msg;
    char* d_msg;

    char h_message[] = "I am pro cuda developer";

    int length = strlen(h_message);

    h_msg = (char*)malloc(length*sizeof(char));
    hipMalloc(&d_msg, length*sizeof(char));

    hipMemcpyToSymbol(HIP_SYMBOL(d_message), h_message, length*sizeof(char));
    hipMemcpyToSymbol(HIP_SYMBOL(d_message), h_message, length * sizeof(char));

    int thread_per_block = length;

    msgRead<<<1, thread_per_block>>>(d_msg);

    hipMemcpy(h_msg, d_msg, length*sizeof(char), hipMemcpyDeviceToHost);

    cout<<h_msg<<endl;

    hipFree(d_msg);
    free(h_msg);



    return 0;
}

