#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void vectorAdd(int* dA, int* dB, int* dC, int size)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;

    if(i<size)
    {
        dC[i] = dA[i] + dB[i];
    }

}

void intializeHostArray(int* hA, int* hB, int size)
{
    for(int i=0; i<size; i++)
    {
        hA[i] = 0;
        hB[i] = i;
    }
}


int main(){

    int size = 1024;

    int* dA, *dB, *dC;
    int* hA, *hB, *hC;


    hA = (int*)malloc(sizeof(int)*size);
    hB = (int*)malloc(sizeof(int)*size);
    hC = (int*)malloc(sizeof(int)*size);

    intializeHostArray(hA, hB, size);

    hipMalloc(&dA, size*sizeof(int));
    hipMalloc(&dB, size*sizeof(int));
    hipMalloc(&dC, size*sizeof(int));

    hipMemcpy(dA, hA, sizeof(int)*size, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, sizeof(int)*size, hipMemcpyHostToDevice);

    int threads = 256;
    int grid = (size + threads-1)/threads;

    vectorAdd<<<grid, threads>>>(dA, dB, dC, size);

    hipMemcpy(hC, dC, sizeof(int)*size, hipMemcpyDeviceToHost);

    for(int i=0; i<size; i++)
    {
        cout<<hC[i]<<endl;
    }

    hipFree(dC);
    free(hC);


    return 0;
}
